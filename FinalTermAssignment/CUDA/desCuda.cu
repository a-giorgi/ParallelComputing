#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : DES_CUDA.cu
 Author      : Andrea Giorgi
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "des.h"
#include <hip/hip_runtime_api.h>
#include <sys/time.h>

//cudaCharset must be equal to charset in des.h
__device__ char cudaCharset[] = {'a','b','c','d','e','f','g','h',
                         'i','j','k','l','m','n','o','p',
                         'q','r','s','t','u','v','w','x',
                         'y','z','A','B','C','D','E','F',
                         'G','H','I','J','K','L','M','N',
                         'O','P','Q','R','S','T','U','V',
                         'W','X','Y','Z','1','2','3','4',
                         '5','6','7','8','9','0','.','/'};

__device__ int cudaNumChars = *(&cudaCharset + 1) - cudaCharset;

// Initial Permutation Table
__device__ int cuda_initial_perm[] = {
        58, 50, 42, 34, 26, 18, 10, 2,
        60, 52, 44, 36, 28, 20, 12, 4,
        62, 54, 46, 38, 30, 22, 14, 6,
        64, 56, 48, 40, 32, 24, 16, 8,
        57, 49, 41, 33, 25, 17,  9, 1,
        59, 51, 43, 35, 27, 19, 11, 3,
        61, 53, 45, 37, 29, 21, 13, 5,
        63, 55, 47, 39, 31, 23, 15, 7
};


// Expansion D-box Table
__device__ int cuda_exp_d[48] = {
        32, 1, 2, 3, 4, 5, 4, 5,
        6, 7, 8, 9, 8, 9, 10, 11,
        12, 13, 12, 13, 14, 15, 16, 17,
        16, 17, 18, 19, 20, 21, 20, 21,
        22, 23, 24, 25, 24, 25, 26, 27,
        28, 29, 28, 29, 30, 31, 32, 1
};

// S-box Table
__device__ int cuda_s[8][4][16] = {
        {
                14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7,
                0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8,
                4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0,
                15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13
        },
        {
                15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10,
                3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5,
                0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15,
                13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9
        },
        {
                10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8,
                13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1,
                13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7,
                1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12
        },
        {
                7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15,
                13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9,
                10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4,
                3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14
        },
        {
                2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9,
                14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6,
                4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14,
                11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3
        },
        {
                12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11,
                10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8,
                9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6,
                4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13
        },
        {
                4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1,
                13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6,
                1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2,
                6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12
        },
        {
                13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7,
                1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2,
                7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8,
                2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11
        }
};

// Straight Permutation Table
__device__ int cuda_per[32] = {
        16, 7, 20,21,
        29, 12, 28, 17,
        1, 15, 23, 26,
        5, 18, 31, 10,
        2, 8, 24, 14,
        32, 27, 3, 9,
        19, 13, 30, 6,
        22, 11, 4, 25
};

// Final Permutation
__device__ int cuda_final_perm[] = {
        40, 8, 48, 16, 56, 24, 64, 32,
        39, 7, 47, 15, 55, 23, 63, 31,
        38, 6, 46, 14, 54, 22, 62, 30,
        37, 5, 45, 13, 53, 21, 61, 29,
        36, 4, 44, 12, 52, 20, 60, 28,
        35, 3, 43, 11, 51, 19, 59, 27,
        34, 2, 42, 10, 50, 18, 58, 26,
        33, 1, 41, 9, 49, 17, 57, 25
};

__device__ unsigned int* cudaDesEncrypt(int blockSize, unsigned int* bin, unsigned int* key, unsigned int* binIP, unsigned int* left,
		unsigned int* right, unsigned int* backupR, unsigned int* xorbin, unsigned int* expanded,
		unsigned int* sBoxResult, unsigned int* ciphertext, unsigned int* sBoxResultFull,unsigned int* sPermResult){
	int length = 64, i,j, round = 0, sbRow, sFullResIndex, sbColumn, value, sResIndex;
	unsigned long int index = blockIdx.x*blockSize+threadIdx.x;

    // Initial Permutation
	for(i=0;i<64;i++){
		binIP[(index*64)+i] = bin[index*64+cuda_initial_perm[i]-1];
	}

	//Now we divide in left and right
	for(i=0; i<length/2; i++){
	    left[(index*32)+i] = binIP[(index*64)+i];
	    right[(index*32)+i] = binIP[(index*64)+i+(length/2)];
	}

	while(round < 16) {
		for (i = 0; i < length / 2; i++) {
		    backupR[(index*32)+i] = right[(index*32)+i];
		}
		sFullResIndex = 0;
		//using the expansionBox
		for(i = 0; i < 48; i++){
		    expanded[(index*48)+i] = right[(index*32)+cuda_exp_d[i]-1];
		}
		//Key XOR Expansion(right)
		for (i = 0; i < 48; i++) {
		    xorbin[(index*48)+i] = (expanded[(index*48)+i]+key[48*round+i]) % 2;
		}
		//SBOX
		for (i = 0; i < 48; i += 6) {
			sbRow = (xorbin[(index*48)+i + 5]*1) + (xorbin[(index*48)+i] * 2); //from 0 to 3
			sbColumn = (xorbin[(index*48)+i + 1]* 8) + (xorbin[(index*48)+i + 2] * 4) + (xorbin[(index*48)+i + 3] *2)
					+ (xorbin[(index*48)+i + 4]*1); //from 0 to 15
			value = cuda_s[i/6][sbRow][sbColumn];
			//converting decimal value inside sBox to binary
			sResIndex = 0;
			while(value>0){
				sBoxResult[(index*4)+sResIndex]= value % 2;
				sResIndex++;
				value/=2;
			}
			while(sResIndex < 4){
				sBoxResult[(index*4)+sResIndex] = 0;
				sResIndex++;
			}
			for (j = sResIndex - 1; j >= 0; j--) {
				sBoxResultFull[(index*48)+sFullResIndex] = sBoxResult[(index*4)+j];
				sFullResIndex++;
			}
		}
		//Now we have the full output from sbox
		for (i = 0; i < 32; i++){
		    sPermResult[(index*32)+i] = sBoxResultFull[(index*48)+cuda_per[i] - 1];
		}
		//Xor with left and sbox output
		for (i = 0; i < 32; i++){
			right[(index*32)+i] = (sPermResult[(index*32)+i] + left[(index*32)+i])%2;
		}
		for (i = 0; i < 32; i++){
		    left[(index*32)+i] = backupR[(index*32)+i];
		}
		round++;
	}

	for(i=0; i<32; i++){
		binIP[(index*64)+i]=right[(index*32)+i];
		binIP[(index*64)+i+32]=left[(index*32)+i];
	}

	for(i=0; i<64; i++){
		ciphertext[(index*64)+i]=binIP[(index*64)+cuda_final_perm[i]-1];
	}

	return ciphertext;

}
//used for debug purpose
__global__ void helloGPU(int blockSize){
	int index = blockIdx.x*blockSize+threadIdx.x;
	printf("Hello from thread: %d\n",index);
}

__device__ char* cudaString2hex(int blockSize, const char input[],unsigned int len, char output[])
{
	unsigned long int index = blockIdx.x*blockSize+threadIdx.x;
    static const char hex_digits[] = "0123456789ABCDEF";
    int count = 0;
    for (int i=0; i<len; ++i) {
        const char c = input[index*8+i];
        output[index*16+count] = (hex_digits[c >> 4]);
        count++;
        output[index*16+count] = (hex_digits[c & 15]);
        count++;

    }
    return output;
}

__device__ unsigned  int* cudaHex2bin(int blockSize, const char input[], int length, unsigned int output[]) //hexadecimal to binary
{
	unsigned long int index = blockIdx.x*blockSize+threadIdx.x;
	const unsigned short int map[64]=
	{
			0, 0, 0, 0,
			0, 0, 0, 1,
			0, 0, 1, 0,
			0, 0, 1, 1,
			0, 1, 0, 0,
			0, 1, 0, 1,
			0, 1, 1, 0,
			0, 1, 1, 1,
			1, 0, 0, 0,
			1, 0, 0, 1,
			1, 0, 1, 0,
			1, 0, 1, 1,
			1, 1, 0, 0,
			1, 1, 0, 1,
			1, 1, 1, 0,
			1, 1, 1, 1
	};
	int arrIndex = 0;

	for(int i = 0; i<length; i++){
		int mapIndex = 0;
		if(input[index*16+i]>='0'&& input[index*16+i]<='9'){
			for(int j = 0; j < 4; j++){
				mapIndex = input[index*16+i] - '0'; //convert char to int maintaining numbers
				output[index*64+arrIndex]= map[mapIndex*4+j];
				arrIndex++;
			}
		}else{
			switch(input[index*16+i]) {
				case 'A':
					mapIndex = 10;
				break;
				case 'B':
					mapIndex = 11;
				break;
				case 'C':
					mapIndex = 12;
				break;
				case 'D':
					mapIndex = 13;
				break;
				case 'E':
					mapIndex = 14;
				break;
				case 'F':
					mapIndex = 15;
				break;
			}
			for(int j = 0; j < 4; j++){
				output[index*64+arrIndex]=map[mapIndex*4+j];
				arrIndex++;
			}
		}
	}

    return output;
}

__device__ unsigned int* generatePassword(int blockSize, char* password, char * hex_password, unsigned int* bin_password, unsigned int length, unsigned long int iteration){
	unsigned long int index = blockIdx.x*blockSize+threadIdx.x;
    for (int j=0; j<(length-1);j++){
    	password[(index*length)+j] = cudaCharset[iteration / (unsigned long int)(pow((double) cudaNumChars ,(double)((length-1)-j)))];
        iteration = iteration % ( unsigned long int)(pow((double) cudaNumChars ,(double)((length-1)-j)));
    }
    password[(index*length)+length-1] = cudaCharset[iteration % cudaNumChars];
    hex_password = cudaString2hex(blockSize, password, length, hex_password);
    //printf("%c%c%c%c%c%c%c%c%c%c%c%c%c%c%c%c\n",hex_password[index*16+0],hex_password[index*16+1],hex_password[index*16+2],hex_password[index*16+3],hex_password[index*16+4],hex_password[index*16+5],hex_password[index*16+6],hex_password[index*16+7],hex_password[index*16+8],hex_password[index*16+9],hex_password[index*16+10],hex_password[index*16+11],hex_password[index*16+12],hex_password[index*16+13],hex_password[index*16+14],hex_password[index*16+15]);
    bin_password = cudaHex2bin(blockSize, hex_password, length*2, bin_password);
    return bin_password;
}

__global__ void findPassword(int blockSize, int numThreads, int passwordLength, unsigned int* targetPassword,
		char* password, char * hex_password, unsigned int* bin_password, unsigned int* key, unsigned int* binIP,
		unsigned int* left, unsigned int* right, unsigned int* backupR, unsigned int* xorbin, unsigned int* expanded,
		unsigned int* sBoxResult, unsigned int* ciphertext, unsigned int* sBoxResultFull, unsigned int* sPermResult){
	unsigned int long const index = blockIdx.x*blockSize+threadIdx.x;
	bool found = false;
	unsigned int l = passwordLength;
	unsigned long int n = cudaNumChars;
	unsigned long int plaintexts = n;
	    for (int i = 1; i < l; i++) {
	    	plaintexts = plaintexts * n;
	    }

	unsigned long int threadIterations = plaintexts/numThreads;
	unsigned long int startIteration = index*threadIterations;
	unsigned long int endIteration = startIteration + threadIterations;
	if(index == plaintexts){
		endIteration = plaintexts;
	}
	//printf("\nnumChars: %d, startIteration: %lu, endIteration: %lu, thread: %lu \n",cudaNumChars,startIteration,endIteration,index);
	for(unsigned long int iteration = startIteration; iteration < endIteration; iteration++){
		//printf("iteration %lu from thread %lu,\n",iteration, index);
		generatePassword(blockSize, password, hex_password, bin_password, 8, iteration);
		cudaDesEncrypt(blockSize, bin_password,key,binIP,left,right,backupR,xorbin,expanded,
			sBoxResult,ciphertext,sBoxResultFull, sPermResult);
		for(int i=0; i<64; i++){
			if(ciphertext[index*64+i] != targetPassword[i]){
				break;
			}
			if( i == 64-1){
				found = true;
			}
		}
		if(found){
			 printf("\nPassword decrypted from thread %lu. The plaintext is: \n", index);
			 for(int i=0; i < 64; i++){
			     printf("%d", bin_password[index*64+i]);
			 }
			 printf("\n");
			 asm("trap;");
			 found = false;
		}
	}
	printf("No results found from thread %d\n",index);
}

int main(){
	int i;
	locale locale;
	std::string numberOfThreads, insertedPassword;
	//char input[16] = {'6','1','6','1','6','1','6','1','6','1','3','6','6','1','5','A'};
	char inputStr[8]; //= {'a','a','a','a','a','a','b','b'};
	char keyText[16] = {'4','1','6','B','6','5','7','9','5','A','5','4','3','3','3','1'};

	//fetching password from user
	getPassword:;
	std::cout << "Write your password (must be 8 characters long):\n";
	std::cin >> insertedPassword;
	if(insertedPassword.length()!=8) {
	    cout<<"The password must be 8 characters long\n";
	    goto getPassword;
	}

	for(int i = 0; i < insertedPassword.length(); i++) {
		for(int j = 0; j < numChars; j++){
			if(insertedPassword[i] == charset[j]){
				break;
			}
			if(j == numChars-1){
				std::cout << "Password contains some illegal characters!\n";
				goto getPassword;
			}
		}
	    inputStr[i] = insertedPassword[i];
	}

	//converting input to hex
	char* input = (char*)malloc(16);
	input = string2hex(inputStr,8,input);

	// converting hex to bin
	char* binary = (char*)malloc(16*4);
	binary = hex2bin(input,16,binary);

	//displaying plaintext
	std::cout<< "Binary password: ";
	for(i =0; i<64;i++){
		std::cout<< binary[i];
	}
	std::cout<< endl;
	int* plaintextInt = new int[64];
	for(i=0;i<64;i++){
		plaintextInt[i] = binary[i]-'0';
	}

	char* keyBin = (char*)malloc(16*4);
	keyBin = hex2bin(keyText,16,keyBin);
	std::cout<< "Key: ";
	for(i=0; i<64;i++){
		std::cout<< keyBin[i];
	}
	std::cout<<endl;

	//generating keys
	unsigned int** key = keyGenerator(keyBin);
    int height = 16;
    int width = 48;

    //placing keys into 1D array
    unsigned int* reducedKey = new unsigned int [height*width];
    for (int h = 0; h < height; h++){
        for (int w = 0; w < width; w++)
        	reducedKey[width * h + w] = key[h][w];
    }

    //Ciphertext target
    unsigned int *ciphertextPassword = (unsigned int*)malloc(64*sizeof(unsigned int*));
    ciphertextPassword = binEncrypt(plaintextInt,reducedKey);
    std::cout<<"Ciphertext (encrypted password): ";
    for(i=0; i<64;i++){
    		std::cout<< ciphertextPassword[i];
    }
    std::cout<<endl;

    unsigned int threads;
    // Allowing user to set threads
    setTreads:;
    std::cout << "Choose the number of threads:\n";
    std::cin >> numberOfThreads;
    for(int i = 0; i < numberOfThreads.length(); i++) {
    	if(!isdigit(numberOfThreads[i], locale)) {
    		std::cout << "Wrong value: you must write an integer!\n";
    		goto setTreads;
    	}
    }
    if(atoi(numberOfThreads.c_str()) != 0)
    	threads = atoi(numberOfThreads.c_str());
    if(threads == 0){
    	threads = 1;
    	std::cout<< "Number of threads set to 1"<<endl;
    }

    /** organizing threads to maximize warps */
    int t, b;
    int blockSize = 32;
    if(threads<=32){
    	b = 1;
    	t = threads;
    	printf("Configuration: <<<%d,%d>>>\n",b,t);
    }else if(threads>32 && (threads%128)==0){
    	b = threads/128;
    	t = 128;
    	blockSize = 128;
    	printf("Configuration: <<<%d,%d>>>\n",b,t);
    }else if(threads>32 && (threads%128)!=0 &&(threads%32)==0){
    	b = threads/32;
    	t = 32;
    	printf("Configuration: <<<%d,%d>>>\n",b,t);
    }else{
    	int q = threads/32;
    	int mod = threads % 32;
    	if(mod>16){
    		threads = (q+1)*32;
    	}else{
    		threads = (q)*32;
    	}
    	std::cout << "Thread fixed to "<< threads <<" to improve performances"<<endl;
    	b = threads/32;
    	t = 32;
    	printf("Configuration: <<<%d,%d>>>\n",b,t);
    }
    /** ------------------ */

    /**CudaMalloc for cudaDesEncrypt */
    unsigned int* binIP;
    unsigned int* left;
    unsigned int* right;
    unsigned int* backupR;
    unsigned int* xorbin;
    unsigned int* expanded;
    unsigned int* sBoxResult;
    unsigned int* ciphertext;
    unsigned int* sBoxResultFull;
    unsigned int* sPermResult;
    unsigned int* cudaKey;

    hipMalloc(&binIP, (threads*64*sizeof(unsigned int)));
    hipMalloc(&left, (threads*64*sizeof(unsigned int))/2);
    hipMalloc(&right, (threads*64*sizeof(unsigned int))/2);
    hipMalloc(&backupR,(threads*64*sizeof(unsigned int))/2);
    hipMalloc(&xorbin, (threads*48*sizeof(unsigned int)));
    hipMalloc(&expanded, (threads*48*sizeof(unsigned int)));
    hipMalloc(&sBoxResult, (threads*4*sizeof(unsigned int)));
    hipMalloc(&ciphertext, (threads*64*sizeof(unsigned int)));
    hipMalloc(&sBoxResultFull,(threads*48*sizeof(unsigned int)));
    hipMalloc(&sPermResult,(threads*32*sizeof(unsigned int)));
    hipMalloc(&cudaKey,(width*height*sizeof(unsigned int)));
    hipMemcpy(cudaKey, reducedKey, width*height* sizeof(unsigned int), hipMemcpyHostToDevice);
    /**-----------------------------*/

    /** CudaMalloc for generatePassword and findPlaintext */
    char* password;
    char* hex_password;
    unsigned int* bin_password;
    unsigned int* targetPassword;

    hipMalloc(&password,(threads*8*sizeof(char)));
    hipMalloc(&hex_password,(threads*8*2*sizeof(char)));
    hipMalloc(&bin_password,(threads*8*2*4*sizeof(unsigned int)));
    hipMalloc(&targetPassword,(64*sizeof(unsigned int)));
    hipMemcpy(targetPassword, ciphertextPassword, 64* sizeof(unsigned int), hipMemcpyHostToDevice);
    /**---------------------------------*/

    struct timeval t0, t1;
    gettimeofday(&t0, NULL);

    int passwordLength = 8;
    unsigned long int possiblePlaintexts = (long)pow((double)numChars,(double)passwordLength);

    printf("Possible plaintexts: %lu\n",possiblePlaintexts);
    printf("Trying every possible combination...\n");
    findPassword<<<b,t>>>(blockSize, threads, 8, targetPassword,password,hex_password,bin_password,
    		cudaKey,binIP,left,right, backupR, xorbin,expanded,sBoxResult,ciphertext,sBoxResultFull,sPermResult);

    hipDeviceSynchronize();

    gettimeofday(&t1, NULL);
    double time = ((t1.tv_sec  - t0.tv_sec) * 1000000u + t1.tv_usec - t0.tv_usec) / 1.e6;
    std::cout<<"Time passed: "<< time <<endl;

    //releasing GPU memory
    hipFree(binIP);
    hipFree(left);
    hipFree(right);
    hipFree(backupR);
    hipFree(xorbin);
    hipFree(expanded);
    hipFree(sBoxResult);
    hipFree(ciphertext);
    hipFree(sBoxResultFull);
    hipFree(sPermResult);
    hipFree(cudaKey);
    hipFree(password);
    hipFree(hex_password);
    hipFree(bin_password);
    hipFree(targetPassword);

    return 0;

}

